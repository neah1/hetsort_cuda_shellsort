#include "hetsort.cuh"

void sortThrust2N(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            int* mainBuffer = gpu.useFirstBuffer ? gpu.buffer1 : gpu.buffer2;
            int* secondaryBuffer = gpu.useFirstBuffer ? gpu.buffer2 : gpu.buffer1;

            if (i == 0) 
                doubleMemcpy(mainBuffer, chunks[i].data(), chunks[i].size(), hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);

            thrustsort(mainBuffer, chunks[i].size(), secondaryBuffer, gpu.bufferSize, gpu.stream1);

            if (i + 1 < chunks.size()) {
                hipStreamSynchronize(gpu.stream1);
                hipMemcpyAsync(chunks[i].data(), mainBuffer, chunks[i].size() * sizeof(int), hipMemcpyDeviceToHost, gpu.stream1);
                hipMemcpyAsync(secondaryBuffer, chunks[i + 1].data(), chunks[i + 1].size() * sizeof(int), hipMemcpyHostToDevice, gpu.streamTmp);
                hipStreamSynchronize(gpu.streamTmp);
            } else {
                doubleMemcpy(chunks[i].data(), mainBuffer, chunks[i].size(), hipMemcpyDeviceToHost, gpu.stream1, gpu.streamTmp);
            }

            gpu.toggleBuffer();
        }
    }
    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);
}

void sortThrust3N(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            int* mainBuffer = gpu.useFirstBuffer ? gpu.buffer1 : gpu.buffer2;
            int* secondaryBuffer = gpu.useFirstBuffer ? gpu.buffer2 : gpu.buffer1;
            hipStream_t mainStream = gpu.useFirstBuffer ? gpu.stream1 : gpu.stream2;
            hipStream_t secondaryStream = gpu.useFirstBuffer ? gpu.stream2 : gpu.stream1;

            if (i == 0) 
                doubleMemcpy(mainBuffer, chunks[i].data(), chunks[i].size(), hipMemcpyHostToDevice, mainStream, gpu.streamTmp);

            thrustsort(mainBuffer, chunks[i].size(), gpu.bufferTmp, gpu.bufferSize, mainStream);

            if ((i > 0)) 
                doubleMemcpy(chunks[i - 1].data(), secondaryBuffer, chunks[i - 1].size(), hipMemcpyDeviceToHost, secondaryStream, gpu.streamTmp);
            
            if (i + 1 < chunks.size()) 
                doubleMemcpy(secondaryBuffer, chunks[i + 1].data(), chunks[i + 1].size(), hipMemcpyHostToDevice, secondaryStream, gpu.streamTmp);
            
            if (i == chunks.size() - 1) 
                hipMemcpyAsync(chunks[i].data(), mainBuffer, chunks[i].size() * sizeof(int), hipMemcpyDeviceToHost, mainStream);

            gpu.toggleBuffer();
        }
    }
    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);
}

void sortThrustInplace(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            if (i == 0) 
                doubleMemcpy(gpu.buffer1, chunks[i].data(), chunks[i].size(), hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);

            thrustsort(gpu.buffer1, chunks[i].size(), gpu.buffer2, gpu.bufferSize, gpu.stream1);

            doubleMemcpy(chunks[i].data(), gpu.buffer1, chunks[i].size(), hipMemcpyDeviceToHost, gpu.stream1, gpu.streamTmp);

            if (i + 1 < chunks.size()) 
                doubleMemcpy(gpu.buffer1, chunks[i + 1].data(), chunks[i + 1].size(), hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);
        }
    }
    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);
}

void sortThrustInplaceMemcpy(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            if (i == 0)
                hipMemcpyAsync(gpu.buffer1, chunks[i].data(), chunks[i].size() * sizeof(int), hipMemcpyHostToDevice, gpu.stream1);

            thrustsort(gpu.buffer1, chunks[i].size(), gpu.buffer2, gpu.bufferSize, gpu.stream1);

            int* nextChunkData = (i + 1 < chunks.size()) ? chunks[i + 1].data() : nullptr;
            size_t nextChunkSize = (i + 1 < chunks.size()) ? chunks[i + 1].size() * sizeof(int) : 0;
            InplaceMemcpy(nextChunkData, gpu.buffer1, chunks[i].data(), nextChunkSize, chunks[i].size() * sizeof(int), gpu.stream1, gpu.streamTmp, 10 * 1024 * 1024);
        }
    }
    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);
}

void sortShell(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            if (i == 0) 
                doubleMemcpy(gpu.buffer1, chunks[i].data(), chunks[i].size(), hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);

            shellsort(gpu.buffer1, chunks[i].size(), gpu.stream1);

            doubleMemcpy(chunks[i].data(), gpu.buffer1, chunks[i].size(), hipMemcpyDeviceToHost, gpu.stream1, gpu.streamTmp);

            if (i + 1 < chunks.size()) 
                doubleMemcpy(gpu.buffer1, chunks[i + 1].data(), chunks[i + 1].size(), hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);
        }
    }
    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);
}

void sortShell2N(std::vector<std::vector<std::vector<int>>>& chunkGroups, std::vector<GPUInfo>& gpus) {
    omp_set_num_threads(gpus.size());
    #pragma omp parallel for
    for (int g = 0; g < static_cast<int>(chunkGroups.size()); ++g) {
        auto& chunks = chunkGroups[g];
        GPUInfo& gpu = gpus[g];
        hipSetDevice(gpu.id);

        for (size_t i = 0; i < chunks.size(); ++i) {
            int* mainBuffer = gpu.useFirstBuffer ? gpu.buffer1 : gpu.buffer2;
            int* secondaryBuffer = gpu.useFirstBuffer ? gpu.buffer2 : gpu.buffer1;
            hipStream_t mainStream = gpu.useFirstBuffer ? gpu.stream1 : gpu.stream2;
            hipStream_t secondaryStream = gpu.useFirstBuffer ? gpu.stream2 : gpu.stream1;

            // Copy the first chunk data to the main GPU buffer
            if (i == 0) 
                doubleMemcpy(mainBuffer, chunks[i].data(), chunks[i].size(), hipMemcpyHostToDevice, mainStream, gpu.streamTmp);

            // Sort the chunk on the main buffer
            shellsort(mainBuffer, chunks[i].size(), mainStream);

            // Copy the sorted chunk back and the next chunk to the secondary buffer
            if ((i > 0)) 
                doubleMemcpy(chunks[i - 1].data(), secondaryBuffer, chunks[i - 1].size(), hipMemcpyDeviceToHost, secondaryStream, gpu.streamTmp);
            
            if (i + 1 < chunks.size()) 
                doubleMemcpy(secondaryBuffer, chunks[i + 1].data(), chunks[i + 1].size(), hipMemcpyHostToDevice, secondaryStream, gpu.streamTmp);

            // Copy the last sorted chunk back
            if (i == chunks.size() - 1) 
                hipMemcpyAsync(chunks[i].data(), mainBuffer, chunks[i].size() * sizeof(int), hipMemcpyDeviceToHost, mainStream);

            // Toggle the buffer for the next chunk
            gpu.toggleBuffer();
        }
    }
    for (auto& gpu : gpus) {
        hipStreamSynchronize(gpu.stream1);
        hipStreamSynchronize(gpu.stream2);
    }
}


std::vector<int> sortKernel(const std::string& method, int* h_inputArray, size_t arraySize, std::vector<GPUInfo>& gpus) {
    std::vector<int> h_outputArray(arraySize);
    GPUInfo& gpu = gpus[0];
    hipSetDevice(gpu.id);

    doubleMemcpy(gpu.buffer1, h_inputArray, arraySize, hipMemcpyHostToDevice, gpu.stream1, gpu.streamTmp);

    if (method == "thrustsortKernel") 
        thrustsort(gpu.buffer1, arraySize, gpu.buffer2, gpu.bufferSize, gpu.stream1);
    else if (method == "shellsortKernel")
        shellsort(gpu.buffer1, arraySize, gpu.stream1);

    doubleMemcpy(h_outputArray.data(), gpu.buffer1, arraySize, hipMemcpyDeviceToHost, gpu.stream1, gpu.streamTmp);

    for (auto& gpu : gpus) hipStreamSynchronize(gpu.stream1);

    return h_outputArray;
}

