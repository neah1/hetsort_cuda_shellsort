#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include "benchmark.h"

// Algorithm parameters
const int seed = 0;
const int warmup = 10;
const int iterations = 10;
const int arraySize = 10'000;
const int increments[] = {1750, 701, 301, 132, 57, 23, 10, 4, 1}; // Increment sequence from Ciura (2001)
// const int increments[] = {5, 3, 1}; // Shell's original sequence
const int numThreads = 256;
const int numBlocks = (arraySize + numThreads - 1) / numThreads;
const int numIncrements = sizeof(increments) / sizeof(increments[0]);
const size_t arrayByteSize = arraySize * sizeof(int);

// Function prototypes
__global__ void parallelShellsort(int *array, int arraySize, int increment);
void warmUpGPU(int *d_array, int *h_inputArray);
void runSort(int *d_array, int *h_inputArray, int *h_outputArray);

int main()
{
    // Allocate and initialize arrays
    int *d_array;
    int *h_inputArray = (int *)malloc(arrayByteSize);
    int *h_outputArray = (int *)malloc(arrayByteSize);
    generateRandomArray(h_inputArray, arraySize, seed);
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_array, arrayByteSize));

    // Run shellsort
    warmUpGPU(d_array, h_inputArray);
    runSort(d_array, h_inputArray, h_outputArray);

    // Free host and device memory
    hipFree(d_array);
    free(h_inputArray);
    free(h_outputArray);

    return 0;
}

void warmUpGPU(int *d_array, int *h_inputArray)
{
    CHECK_CUDA_ERROR(hipMemcpy(d_array, h_inputArray, arrayByteSize, hipMemcpyHostToDevice));
    for (int i = 0; i < warmup; i++)
    {
        parallelShellsort<<<numBlocks, numThreads>>>(d_array, arraySize, 1);
        hipDeviceSynchronize();
    }
}

void runSort(int *d_array, int *h_inputArray, int *h_outputArray)
{
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalTime = 0.0f;
    for (int i = 0; i < iterations; i++)
    {
        // Copy unsorted array to device
        CHECK_CUDA_ERROR(hipMemcpy(d_array, h_inputArray, arrayByteSize, hipMemcpyHostToDevice));

        // Start recording
        hipEventRecord(start, NULL);
        nvtxRangePush("Shellsort");

        // Run parallel shell-sort for each increment
        for (int j = 0; j < numIncrements; j++)
        {
            parallelShellsort<<<numBlocks, numThreads>>>(d_array, arraySize, increments[j]);
            hipDeviceSynchronize();
        }

        // Stop recording
        nvtxRangePop();
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);

        // Calculate elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        totalTime += milliseconds;

        // Copy sorted array back to host and verify
        CHECK_CUDA_ERROR(hipMemcpy(h_outputArray, d_array, arrayByteSize, hipMemcpyDeviceToHost));
        if (!checkArraySorted(h_inputArray, h_outputArray, arraySize))
        {
            fprintf(stderr, "Error: Array not sorted correctly\n");
            exit(EXIT_FAILURE);
        }
    }

    // Compute and print the average time per iteration
    float avgTime = totalTime / iterations;
    printf("Average Time for Sorting: %f ms\n", avgTime);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}