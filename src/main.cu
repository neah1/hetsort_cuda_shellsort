#include "hetsort.cuh"

// Algorithm parameters
std::string method = "shellsort2N";
std::string distribution = "uniform";
size_t arraySize = 1'000'000'000;
size_t deviceMemory = 1'000;

// Algorithm parameters
const int seed = 42;
const int warmup = 0;
const int iterations = 1;

typedef void CUDASorter(std::vector<std::vector<std::vector<int>>>&, std::vector<GPUInfo>&);

std::vector<int> runSort(CUDASorter cudaSorter, int* h_inputArray, size_t arraySize, size_t chunkSize, std::vector<GPUInfo>& gpus) {
    nvtxRangePush("Split array phase");
    if (method.find("thrust") == 0) chunkSize = chunkSize * 0.80;
    std::vector<std::vector<std::vector<int>>> chunkGroups = splitArray(h_inputArray, arraySize, chunkSize, gpus);
    nvtxRangePop();

    nvtxRangePush("Kernel phase");
    cudaSorter(chunkGroups, gpus);
    nvtxRangePop();

    nvtxRangePush("Merge phase");
    std::vector<int> h_outputArray = multiWayMerge(chunkGroups);
    nvtxRangePop();

    return h_outputArray;
}

void benchmark(CUDASorter cudaSorter, int* h_inputArray, size_t arraySize, size_t chunkSize, std::vector<GPUInfo>& gpus) {
    // Count the number of elements in the input array
    std::unordered_map<int, int> originalCounts = countElements(h_inputArray, arraySize);

    // Warmup the GPU
    for (int i = 0; i < warmup; i++) runSort(cudaSorter, h_inputArray, arraySize, chunkSize, gpus);

    for (int i = 0; i < iterations; i++) {
        // Start timing
        auto start = std::chrono::high_resolution_clock::now();
        
        nvtxRangePush("HETSort algorithm");
        std::vector<int> h_outputArray = runSort(cudaSorter, h_inputArray, arraySize, chunkSize, gpus);
        nvtxRangePop();

        // Stop timing
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
        printf("Iteration %d: %f ms\n", i, duration);

        // Check if the array is sorted correctly
        if (!checkArraySorted(h_outputArray.data(), originalCounts, arraySize)) {
            fprintf(stderr, "Error (%s): Array not sorted correctly\n", method.c_str());
            exit(EXIT_FAILURE);
        }
    }
}

CUDASorter* selectSortingMethod(const std::string& method, size_t& bufferCount) {
    CUDASorter* cudaSorter = nullptr;
    if (method == "thrustsort2N") {
        cudaSorter = sortThrust2N;
        bufferCount = 2;
    } else if (method == "thrustsort3N") {
        cudaSorter = sortThrust3N;
        bufferCount = 3;
    } else if (method == "thrustsortInplace") {
        cudaSorter = sortThrustInplace;
        bufferCount = 2;
    } else if (method == "shellsort") {
        cudaSorter = sortShell;
        bufferCount = 1;
    } else if (method == "shellsort2N") {
        cudaSorter = sortShell2N;
        bufferCount = 2;
    } else {
        std::cerr << "Invalid sorting method.\n";
        exit(EXIT_FAILURE);
    }
    return cudaSorter;
}

void runSortingAlgorithm(const std::string& method, const std::string& distribution, size_t arraySize, size_t deviceMemory) {
    // Select sorting method
    size_t bufferCount;
    CUDASorter* cudaSorter = selectSortingMethod(method, bufferCount);

    // Calculate chunk size
    deviceMemory = deviceMemory * 1024 * 1024;
    size_t chunkSize = (deviceMemory / bufferCount) / sizeof(int);

    nvtxRangePush("Get GPU information");
    std::vector<GPUInfo> gpus = getGPUsInfo(deviceMemory, bufferCount);
    nvtxRangePop();

    nvtxRangePush("Generate array distribution");
    int* h_inputArray;
    hipHostMalloc((void**)&h_inputArray, arraySize * sizeof(int));
    generateRandomArray(h_inputArray, arraySize, seed, distribution);
    nvtxRangePop();

    // Run sorting algorithm
    benchmark(cudaSorter, h_inputArray, arraySize, chunkSize, gpus);

    // Clean up
    hipHostFree(h_inputArray);
}
void fullBenchmark() {
    std::vector<std::string> methods = {"thrustsort2N", "thrustsort3N", "thrustsortInplace", "shellsort", "shellsort2N"};
    std::vector<std::string> distributions = {"uniform", "normal", "sorted", "reverse_sorted", "nearly_sorted"};
    std::vector<size_t> arraySizes = {1'000'000, 10'000'000};
    std::vector<size_t> deviceMemories = {500, 1000, 2000};

    // Loop over each combination of parameters
    for (const auto& method : methods) {
        for (const auto& distribution : distributions) {
            for (const auto& arraySize : arraySizes) {
                for (const auto& deviceMemory : deviceMemories) {
                    runSortingAlgorithm(method, distribution, arraySize, deviceMemory);
                }
            }
        }
    }
}

int main(int argc, char* argv[]) {
    method = (argc > 1) ? argv[1] : method;
    distribution = (argc > 2) ? argv[2] : distribution;
    arraySize = (argc > 3) ? std::atoi(argv[3]) : arraySize;
    deviceMemory = (argc > 4) ? std::atoi(argv[4]) : deviceMemory;
    printf("Method: %s. Distribution: %s. Array size: %zu. Array byte size: %zu MB. Device memory: %zu MB. Warmup: %d. Iterations: %d.\n", 
        method.c_str(), distribution.c_str(), arraySize, arraySize * sizeof(int) / (1024 * 1024), deviceMemory, warmup, iterations);

    runSortingAlgorithm(method, distribution, arraySize, deviceMemory);
    return 0;
}