#include "hip/hip_runtime.h"
#include "hetsort.cuh"

GPUInfo::GPUInfo(int id, size_t bufferSize, bool buffers2N)
    : id(id), bufferSize(bufferSize), buffers2N(buffers2N), useFirstBuffer(true) {
    hipSetDevice(id);
    hipStreamCreate(&stream1);
    hipStreamCreate(&streamTmp);
    hipMalloc(&buffer1, bufferSize);
    if (buffers2N) {
        hipStreamCreate(&stream2);
        hipMalloc(&buffer2, bufferSize);
    }
}

GPUInfo::~GPUInfo() {
    hipSetDevice(id);
    hipStreamDestroy(stream1);
    hipStreamDestroy(streamTmp);
    hipFree(buffer1);
    if (buffers2N) {
        hipStreamDestroy(stream2);
        hipFree(buffer2);
    }
}

void GPUInfo::toggleBuffer() {
    if (buffers2N) useFirstBuffer = !useFirstBuffer;
}

std::vector<GPUInfo> getGPUsInfo(size_t bufferSize, bool buffers2N) {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::vector<GPUInfo> gpus;
    gpus.reserve(numGPUs);
    size_t requiredMem = buffers2N ? bufferSize * 2 : bufferSize;

    #pragma omp parallel for
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        #pragma omp critical
        {
            if (freeMem >= requiredMem) {
                gpus.emplace_back(i, bufferSize, buffers2N);
                printf("GPU %d: %zu MB free, %zu MB total\n", i, freeMem / (1024 * 1024), totalMem / (1024 * 1024));
            } else {
                printf("GPU %d: %zu MB free, %zu MB total - Skipped\n", i, freeMem / (1024 * 1024), totalMem / (1024 * 1024));
            }
        }

    }
    printf("GPUs available: %zu\n", gpus.size());
    return gpus;
}

std::vector<std::vector<std::vector<int>>> splitArray(int* unsortedArray, size_t arraySize, size_t bufferSize, std::vector<GPUInfo>& gpus) {
    std::vector<std::vector<int>> chunks;
    
    size_t chunkElementCount = bufferSize / sizeof(int);
    size_t numChunks = arraySize / chunkElementCount + (arraySize % chunkElementCount != 0);
    chunks.reserve(numChunks);

    printf("Splitting array into %zu chunks\n", numChunks);

    // Split the array into chunks
    for (size_t i = 0; i < numChunks; ++i) {
        size_t startIdx = i * chunkElementCount;
        size_t endIdx = std::min(startIdx + chunkElementCount, arraySize);
        chunks.emplace_back(unsortedArray + startIdx, unsortedArray + endIdx);
    }

    // Assign chunks to GPUs
    std::vector<std::vector<std::vector<int>>> chunkGroups(gpus.size());
    for (size_t i = 0; i < chunks.size(); ++i) {
        size_t gpuIndex = i % gpus.size();
        chunkGroups[gpuIndex].push_back(chunks[i]);
    }

    return chunkGroups;
}

std::vector<int> multiWayMerge(const std::vector<std::vector<std::vector<int>>>& chunkGroups) {
    // Prepare a vector of sequences for the multi-way merge from chunk groups
    std::vector<std::pair<int*, int*>> sequences;
    for (const auto& group : chunkGroups) {
        for (const auto& chunk : group) {
            if (!chunk.empty()) sequences.emplace_back(const_cast<int*>(chunk.data()), const_cast<int*>(chunk.data()) + chunk.size());
        }
    }

    // Calculate the total size for the merged result
    size_t total_size = 0;
    for (const auto& seq : sequences) total_size += std::distance(seq.first, seq.second);
    std::vector<int> merged_result(total_size);

    // Perform the multiway merge
    __gnu_parallel::multiway_merge(sequences.begin(), sequences.end(), merged_result.begin(), total_size, std::less<int>());
    return merged_result;
}
