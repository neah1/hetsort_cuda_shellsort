#include "hip/hip_runtime.h"
#include "hetsort.cuh"

GPUInfo::GPUInfo(int id, size_t bufferSize, size_t bufferCount)
    : id(id), bufferSize(bufferSize), bufferCount(bufferCount), useFirstBuffer(true) {
    hipSetDevice(id);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&streamTmp);
    hipMalloc(&buffer1, bufferSize);
    if (bufferCount > 1) hipMalloc(&buffer2, bufferSize);
    if (bufferCount > 2) hipMalloc(&bufferTmp, bufferSize);
}

GPUInfo::~GPUInfo() {
    hipSetDevice(id);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(streamTmp);
    hipFree(buffer1);
    if (bufferCount > 1) hipFree(buffer2);
    if (bufferCount > 2) hipFree(bufferTmp);
}

void GPUInfo::toggleBuffer() {
    useFirstBuffer = !useFirstBuffer;
}

std::vector<GPUInfo> getGPUsInfo(size_t deviceMemory, size_t bufferCount) {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::vector<GPUInfo> gpus;
    gpus.reserve(numGPUs);
    size_t bufferSize = deviceMemory / bufferCount;

    #pragma omp parallel for
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        #pragma omp critical
        {
            if (deviceMemory <= freeMem) {
                gpus.emplace_back(i, bufferSize, bufferCount);
                std::cout << "GPU " << i << ": " << freeMem / (1024 * 1024) << " MB free, " << totalMem / (1024 * 1024) << " MB total\n";
            } else {
                std::cout << "GPU " << i << ": " << freeMem / (1024 * 1024) << " MB free, " << totalMem / (1024 * 1024) << " MB total - Skipped\n";
            }
        }

    }
    std::cout << "GPUs available: " << gpus.size() << "\n";
    return gpus;
}

std::vector<std::vector<std::vector<int>>> splitArray(int* unsortedArray, size_t arraySize, size_t chunkSize, std::vector<GPUInfo>& gpus) {
    std::vector<std::vector<int>> chunks;
    size_t numChunks = arraySize / chunkSize + (arraySize % chunkSize != 0);
    std::cout << "Number of chunks: " << numChunks << "\n";
    
    // Split the array into chunks
    chunks.reserve(numChunks);
    for (size_t i = 0; i < numChunks; ++i) {
        size_t startIdx = i * chunkSize;
        size_t endIdx = std::min(startIdx + chunkSize, arraySize);
        chunks.emplace_back(unsortedArray + startIdx, unsortedArray + endIdx);
    }

    // Assign chunks to GPUs
    std::vector<std::vector<std::vector<int>>> chunkGroups(gpus.size());
    for (size_t i = 0; i < chunks.size(); ++i) {
        size_t gpuIndex = i % gpus.size();
        chunkGroups[gpuIndex].push_back(chunks[i]);
    }

    return chunkGroups;
}

std::vector<int> multiWayMerge(const std::vector<std::vector<std::vector<int>>>& chunkGroups) {
    // Prepare a vector of sequences for the multi-way merge from chunk groups
    std::vector<std::pair<int*, int*>> sequences;
    for (const auto& group : chunkGroups) {
        for (const auto& chunk : group) {
            if (!chunk.empty()) sequences.emplace_back(const_cast<int*>(chunk.data()), const_cast<int*>(chunk.data()) + chunk.size());
        }
    }

    // Calculate the total size for the merged result
    size_t total_size = 0;
    for (const auto& seq : sequences) total_size += std::distance(seq.first, seq.second);
    std::vector<int> merged_result(total_size);

    // Perform the multiway merge
    __gnu_parallel::multiway_merge(sequences.begin(), sequences.end(), merged_result.begin(), total_size, std::less<int>());
    return merged_result;
}
