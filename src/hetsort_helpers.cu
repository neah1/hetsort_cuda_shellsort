#include "hip/hip_runtime.h"
#include "hetsort.cuh"

GPUInfo::GPUInfo(int id, size_t bufferSize, bool buffers2N)
    : id(id), bufferSize(bufferSize), buffers2N(buffers2N), useFirstBuffer(true) {
    hipSetDevice(id);
    hipStreamCreate(&stream1);
    hipStreamCreate(&streamTmp);
    hipMalloc(&buffer1, bufferSize);
    if (buffers2N) {
        hipStreamCreate(&stream2);
        hipMalloc(&buffer2, bufferSize);
    }
}

GPUInfo::~GPUInfo() {
    hipSetDevice(id);
    hipStreamDestroy(stream1);
    hipStreamDestroy(streamTmp);
    hipFree(buffer1);
    if (buffers2N) {
        hipStreamDestroy(stream2);
        hipFree(buffer2);
    }
}

void GPUInfo::toggleBuffer() {
    if (buffers2N) useFirstBuffer = !useFirstBuffer;
}

std::vector<GPUInfo> getGPUsInfo(size_t bufferSize, bool buffers2N) {
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    std::vector<GPUInfo> gpus;
    gpus.reserve(numGPUs);
    size_t requiredMem = buffers2N ? bufferSize * 2 : bufferSize;

    #pragma omp parallel for
    for (int i = 0; i < numGPUs; ++i) {
        hipSetDevice(i);
        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        #pragma omp critical
        {
            if (freeMem >= requiredMem) {
                gpus.emplace_back(i, bufferSize, buffers2N);
                printf("GPU %d: %zu MB free, %zu MB total\n", i, freeMem / (1024 * 1024), totalMem / (1024 * 1024));
            } else {
                printf("GPU %d: %zu MB free, %zu MB total - Skipped\n", i, freeMem / (1024 * 1024), totalMem / (1024 * 1024));
            }
        }

    }
    printf("GPUs available: %zu\n", gpus.size());
    return gpus;
}

std::vector<std::vector<std::vector<int>>> splitArray(int* unsortedArray, size_t arraySize, size_t bufferSize, std::vector<GPUInfo>& gpus) {
    std::vector<std::vector<int>> chunks;
    
    size_t numChunks = arraySize / bufferSize + (arraySize % bufferSize != 0);
    chunks.reserve(numChunks);

    // Split the array into chunks
    for (size_t i = 0; i < numChunks; ++i) {
        size_t startIdx = i * bufferSize;
        size_t endIdx = std::min(startIdx + bufferSize, arraySize);
        chunks.emplace_back(unsortedArray + startIdx, unsortedArray + endIdx);
    }

    size_t numGPUs = gpus.size();
    std::vector<std::vector<std::vector<int>>> chunkGroups(numGPUs);

    // Assign chunks to GPUs
    for (size_t i = 0; i < chunks.size(); ++i) {
        size_t gpuIndex = i % numGPUs;
        chunkGroups[gpuIndex].push_back(chunks[i]);
    }

    return chunkGroups;
}

std::vector<int> multiWayMerge(const std::vector<std::vector<std::vector<int>>>& chunkGroups) {
    // Prepare a vector of sequences for the multi-way merge from chunk groups
    std::vector<std::pair<int*, int*>> sequences;
    for (const auto& group : chunkGroups) {
        for (const auto& chunk : group) {
            if (!chunk.empty()) sequences.emplace_back(const_cast<int*>(chunk.data()), const_cast<int*>(chunk.data()) + chunk.size());
        }
    }

    // Calculate the total size for the merged result
    size_t total_size = 0;
    for (const auto& seq : sequences) total_size += std::distance(seq.first, seq.second);
    std::vector<int> merged_result(total_size);

    // Perform the multiway merge
    __gnu_parallel::multiway_merge(sequences.begin(), sequences.end(), merged_result.begin(), total_size, std::less<int>());
    return merged_result;
}

int main(int argc, char* argv[]) {
    int seed = 42;
    size_t arraySize = (argc > 1) ? std::atoi(argv[1]) : 1'000'000;
    size_t bufferSize = (argc > 2) ? std::atoi(argv[2]) : 200;
    bool buffers2N = (argc > 3) ? std::atoi(argv[3]) : true;
    printf("Array size: %zu. Buffer size: %zu MB. Double buffer: %s\n", arraySize, bufferSize, buffers2N ? "true" : "false");

    // Allocate and initialize arrays
    int* h_inputArray = (int*)malloc(arraySize * sizeof(int));
    generateRandomArray(h_inputArray, arraySize, seed);
    std::unordered_map<int, int> originalCounts = countElements(h_inputArray, arraySize);

    // Get GPU information
    bufferSize = bufferSize * 1024 * 1024;
    std::vector<GPUInfo> gpus = getGPUsInfo(bufferSize, buffers2N);

    // Split the array into chunks based on GPU memory availability
    std::vector<std::vector<std::vector<int>>> chunkGroups = splitArray(h_inputArray, arraySize, bufferSize, gpus);

    // Sort each chunk on the GPU
    sortChunkGroups(chunkGroups, gpus);

    // Check if each chunk is sorted correctly
    if (checkChunkGroupsSorted(originalCounts, chunkGroups)) printf("Chunks are sorted correctly\n");

    // Perform multi-way merge
    std::vector<int> merged_result = multiWayMerge(chunkGroups);

    // Check if the merged array is sorted correctly
    if (checkArraySorted(merged_result.data(), originalCounts, arraySize)) printf("Array is sorted correctly\n");

    // Clean up
    free(h_inputArray);
    return 0;
}