#include "hip/hip_runtime.h"
#include "hetsort.cuh"

void doubleMemcpy(int* dest_array, const int* source_array, size_t arraySize, hipMemcpyKind memcpyMode, hipStream_t stream1, hipStream_t stream2) {
    size_t halfSize = arraySize / 2;
    size_t halfByteSize = halfSize * sizeof(int);
    size_t arrayByteSize = arraySize * sizeof(int);

    // Wait for the sorting stream to complete
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));

    // Start async copy of the first half of the array
    CHECK_CUDA_ERROR(hipMemcpyAsync(dest_array, source_array, halfByteSize, memcpyMode, stream1));

    // Start async copy of the second half of the array
    CHECK_CUDA_ERROR(hipMemcpyAsync(dest_array + halfSize, source_array + halfSize, arrayByteSize - halfByteSize, memcpyMode, stream2));

    // Wait for tmp stream to complete
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));
}

void InplaceMemcpy(int* htod_source, int* dtoh_source, int* dtoh_dest, size_t num_bytes_htod, size_t num_bytes_dtoh,
                   hipStream_t htod_stream, hipStream_t dtoh_stream, size_t block_size) {

  if (dtoh_dest == nullptr && htod_source == nullptr) return;

  size_t num_bytes;
  if (dtoh_dest == nullptr) {
    num_bytes = num_bytes_htod;
    block_size = num_bytes;
  } else if (htod_source == nullptr) {
    num_bytes = num_bytes_dtoh;
    block_size = num_bytes;
  } else {
    num_bytes = std::min(num_bytes_htod, num_bytes_dtoh);
    block_size = std::min(block_size, num_bytes);
  }

  size_t bytes_dtoh = 0;

  if (dtoh_dest != nullptr) {
    CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest, dtoh_source, block_size, hipMemcpyDeviceToHost, dtoh_stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));
  }
  bytes_dtoh += block_size;

  while (bytes_dtoh < num_bytes && dtoh_dest != nullptr && htod_source != nullptr) {
    CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + bytes_dtoh - block_size, htod_source + bytes_dtoh - block_size, block_size, hipMemcpyHostToDevice, htod_stream));

    block_size = std::min(block_size, num_bytes - bytes_dtoh);

    CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest + bytes_dtoh, dtoh_source + bytes_dtoh, block_size, hipMemcpyDeviceToHost, dtoh_stream));

    CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));

    bytes_dtoh += block_size;
  }

  if (htod_source != nullptr) {
    CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + bytes_dtoh - block_size, htod_source + bytes_dtoh - block_size, block_size, hipMemcpyHostToDevice, htod_stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));
  }

  if (num_bytes_htod != num_bytes_dtoh && dtoh_dest != nullptr && htod_source != nullptr) {
    if (num_bytes_htod > num_bytes_dtoh) {
      CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + num_bytes, htod_source + num_bytes, num_bytes_htod - num_bytes, hipMemcpyHostToDevice, htod_stream));
      CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));

    } else if (num_bytes_dtoh > num_bytes_htod) {
      CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest + num_bytes, dtoh_source + num_bytes, num_bytes_dtoh - num_bytes, hipMemcpyDeviceToHost, dtoh_stream));
      CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));
    }
  }
}