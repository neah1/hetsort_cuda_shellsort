#include "hip/hip_runtime.h"
#include "hetsort.cuh"

void doubleMemcpy(int* d_array, const int* h_array, size_t arraySize, hipStream_t stream1, hipStream_t stream2) {
    size_t halfSize = arraySize / 2;
    size_t halfSizeBytes = halfSize * sizeof(int);
    size_t arraySizeBytes = arraySize * sizeof(int);

    // Start async copy of the first half of the array
    hipMemcpyAsync(d_array, h_array, halfSizeBytes, hipMemcpyHostToDevice, stream1);

    // Start async copy of the second half of the array
    hipMemcpyAsync(d_array + halfSize, h_array + halfSize, arraySizeBytes - halfSizeBytes, hipMemcpyHostToDevice, stream2);

    // Wait for both streams to complete
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
}

void InplaceMemcpy(int* htod_source, int* dtoh_source, int* dtoh_dest, size_t num_bytes_htod, size_t num_bytes_dtoh,
                   hipStream_t htod_stream, hipStream_t dtoh_stream, size_t block_size) {
    if (dtoh_dest == nullptr && htod_source == nullptr) {
        return;
    }

    size_t num_bytes;

    if (dtoh_dest == nullptr) {
        num_bytes = num_bytes_htod;
        block_size = num_bytes;
    } else if (htod_source == nullptr) {
        num_bytes = num_bytes_dtoh;
        block_size = num_bytes;
    } else {
        num_bytes = std::min(num_bytes_htod, num_bytes_dtoh);
        block_size = std::min(block_size, num_bytes);
    }

    size_t bytes_dtoh = 0;

    if (dtoh_dest != nullptr) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest, dtoh_source, block_size, hipMemcpyDeviceToHost, dtoh_stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));
    }
    bytes_dtoh += block_size;

    while (bytes_dtoh < num_bytes && dtoh_dest != nullptr && htod_source != nullptr) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + bytes_dtoh - block_size, htod_source + bytes_dtoh - block_size,
                                         block_size, hipMemcpyHostToDevice, htod_stream));

        block_size = std::min(block_size, num_bytes - bytes_dtoh);

        CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest + bytes_dtoh, dtoh_source + bytes_dtoh, block_size, hipMemcpyDeviceToHost,
                                         dtoh_stream));

        CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));

        bytes_dtoh += block_size;
    }

    if (htod_source != nullptr) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + bytes_dtoh - block_size, htod_source + bytes_dtoh - block_size,
                                         block_size, hipMemcpyHostToDevice, htod_stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));
    }

    if (num_bytes_htod != num_bytes_dtoh && dtoh_dest != nullptr && htod_source != nullptr) {
        if (num_bytes_htod > num_bytes_dtoh) {
            CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_source + num_bytes, htod_source + num_bytes, num_bytes_htod - num_bytes,
                                             hipMemcpyHostToDevice, htod_stream));
            CHECK_CUDA_ERROR(hipStreamSynchronize(htod_stream));

        } else if (num_bytes_dtoh > num_bytes_htod) {
            CHECK_CUDA_ERROR(hipMemcpyAsync(dtoh_dest + num_bytes, dtoh_source + num_bytes, num_bytes_dtoh - num_bytes,
                                             hipMemcpyDeviceToHost, dtoh_stream));
            CHECK_CUDA_ERROR(hipStreamSynchronize(dtoh_stream));
        }
    }
}